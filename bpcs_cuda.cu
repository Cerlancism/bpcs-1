#include "hip/hip_runtime.h"
#include "units.h" // for uintN_t typedefs
#include "config.h" // for N_BITPLANES, N_CHANNELS
#define SET_THREAD_COORD(row, x) int row = (blockIdx.y * blockDim.y) + threadIdx.y
#define SET_THREAD_COORDS() SET_THREAD_COORD(row, x); SET_THREAD_COORD(col, y)

#include <cstdio> // for printf

/* See https://en.wikipedia.org/wiki/CUDA for variious hardware-specific constraints. */


/* Device */

__global__ void gpu_rgb2cgc(
    int n_pixels, uint8_t* arr
  #ifdef DEBUG
    , uint32_t* d_n
  #endif
){
    /*
        Inputs:
            row_ptrs:
                Matrix of 8-bit pixels
    */
    int indx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (indx < n_pixels){
        // Order is because most images are landscape, therefore - and this is just a totally uneducated guess - I would expect the height condition to fail more often
        arr[indx] ^= (arr[indx] >> 1);
      #ifdef DEBUG
        atomicAdd(d_n, 1);
      #endif
    }
}
/*
inline __device__ uint8_t gpu_calculate_grid_complexity(uint8_t bitgrid[81]){
    uint8_t c = 0;
    for (int i=0;  i < 9*9 - 1;  ++i)
        if (i % 9 != 9 - 1)
            c += bitgrid[i] ^ bitgrid[i+1];
    for (int i=0;  i < 9;  ++i)
        for (int j=0;  j < 8;  ++j)
            c += bitgrid[9*j + i] ^ bitgrid[9*(j+1) + i];
    return c;
}
*/
/*
inline __device__ void gpu_conjugate_grid(int row, int col, uint8_t** row_ptrs){
    for (int i=0; i<8; ++i)
        row_ptrs[row][col] &= (row + col) & 1;
}
*/
inline __device__ uint8_t gpu_calculate_grid_complexity(int row, int col, uint32_t w, uint8_t*& arr){
    uint8_t c = 0;
    for (int j=0; j<8; ++j)
        for (int i=0; i<9; ++i)
            c  +=  arr[w*(row + j) + (col + i)]  ^  arr[w*(row + j + 1) + (col + i)];
    for (int j=0; j<9; ++j)
        for (int i=0; i<8; ++i)
            c  +=  arr[w*(row + j) + (col + i)]  ^  arr[w*(row + j)     + (col + i + 1)];
    return c;
}

__global__ void gpu_extract_bytes(
    uint8_t t, uint32_t w, uint32_t h, uint32_t n_hztl_grids, uint32_t n_vtcl_grids, uint8_t*& arr, uint8_t*& extraced_bytes
  #ifdef DEBUG
    , uint32_t* d_ns
  #endif
){
    /*
        Inputs:
            t
                aka complexity_threshold
            arr
                Channel (i.e. byteplane) matrix, flattened
    */
    SET_THREAD_COORD(row, y);
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;
    
    if (col < n_hztl_grids && row < n_vtcl_grids){
        for (int j=0; j<N_BITPLANES; ++j){
            
            if (gpu_calculate_grid_complexity(row, col, w, arr) >= t){
                extraced_bytes[11*(w*row + col) + 0] = 1;
                for (int i=1; i<11; ++i){
                    extraced_bytes[11*(w*row + col) + i] = 0;
                    for (int k=0; k<8; ++k){
                        extraced_bytes[11*(w*row + col) + i] *= 2;
                        extraced_bytes[11*(w*row + col) + i] |= (arr[w*row + col + k] >> j) & 1;
                    }
                }
                
            }
        }
      #ifdef DEBUG
        atomicAdd(&d_ns[blockIdx.y*blockDim.x + blockIdx.x], 1);
      #endif
    }
}



/* Host */

void rgb2cgc(uint32_t n_pixels, uint8_t*& host_img_data){
    int n_grids;
    int n_blocks;
    //hipOccupancyMaxPotentialBlockSize(&n_grids, &n_blocks, gpu_rgb2cgc, 0, w*h);
    n_grids=n_pixels/1024; n_blocks=1024;
    
  #ifdef DEBUG
    printf("rgb2cgc\t%d grids\t%d threads/grid\n", n_grids, n_blocks); // tmp
    
    uint32_t n_threads_completed = 0;
    uint32_t* d_n;
    hipMalloc(&d_n, sizeof(uint32_t));
    hipMemcpy(d_n, &n_threads_completed, sizeof(uint32_t), hipMemcpyHostToDevice);
  #endif
    
    uint8_t* arr;
    hipMalloc(&arr, n_pixels);
    hipMemcpy(arr, host_img_data, n_pixels, hipMemcpyHostToDevice);
    
    gpu_rgb2cgc<<<n_grids, n_blocks>>>(
        n_pixels, arr
      #ifdef DEBUG
        , d_n
      #endif
    );
    hipDeviceSynchronize();
    
  #ifdef DEBUG
    hipMemcpy(&n_threads_completed, d_n, sizeof(uint32_t), hipMemcpyDeviceToHost);
    printf("n_threads_completed\t%d\n", n_threads_completed);
  #endif
    
    hipMemcpy(host_img_data, arr, n_pixels, hipMemcpyDeviceToHost);
    hipFree(arr);
}

void extract_bytes(uint8_t t, uint32_t w, uint32_t h, uint32_t n_hztl_grids, uint32_t n_vtcl_grids, uint8_t*& host_img_data, uint8_t*& host_extraced_bytes){
    /*
        Inputs:
            host_extraced_bytes
                Empty array to write to
    */
    //hipOccupancyMaxPotentialBlockSize(&dim_grids, &n_blocks, gpu_rgb2cgc, 0, n_hztl_grids*n_vtcl_grids);
    dim3 dim_grids(n_hztl_grids/32, n_vtcl_grids/32);
    dim3 n_blocks(32, 32);
    // Memory constrains -> smaller blocks?
    
  #ifdef DEBUG
    printf("extract_bytes\n"); // tmp
    
    auto n_grids = (n_hztl_grids/32) * (n_vtcl_grids/32);
    
    uint32_t h_ns[n_grids];
    for (auto i=0;  i<n_grids;  ++i)
        h_ns[i] = 0;
    
    uint32_t* d_ns;
    hipMalloc(&d_ns, sizeof(uint32_t) * n_grids);
    hipMemcpy(d_ns, h_ns, sizeof(uint32_t) * n_grids, hipMemcpyHostToDevice);
  #endif
    
    uint8_t* arr;
    hipMalloc(&arr, w*h*sizeof(uint8_t));
    hipMemcpy(arr, host_img_data, h*w*sizeof(uint8_t), hipMemcpyHostToDevice);
    
    auto n_extracted_bytes = n_hztl_grids*n_vtcl_grids*11*sizeof(uint8_t);
    uint8_t* extraced_bytes;
    hipMalloc(&extraced_bytes, n_extracted_bytes);
    
  #ifdef TESTS
    uint8_t extracted_bytes_zeros[n_hztl_grids*n_vtcl_grids*11];
    for (auto i=0; i<n_hztl_grids*n_vtcl_grids*11; ++i)
        extracted_bytes_zeros[i] = 0;
    hipMemcpy(extraced_bytes, extracted_bytes_zeros, n_hztl_grids*n_vtcl_grids*11*sizeof(uint8_t), hipMemcpyHostToDevice);
  #endif
    
    gpu_extract_bytes<<<dim_grids, n_blocks>>>(
        t, w, h, n_hztl_grids, n_vtcl_grids, arr, extraced_bytes
      #ifdef DEBUG
        , d_ns
      #endif
    );
    hipDeviceSynchronize();
    
  #ifdef DEBUG
    uint32_t n_threads_completed = 0;
    
    hipMemcpy(h_ns, d_ns, sizeof(uint32_t) * n_grids, hipMemcpyDeviceToHost);
    hipFree(d_ns);
    
    for (auto i=0;  i<n_grids;  ++i){
        printf("n_threads_completed in block %d\t%d\n", i, h_ns[i]);
        n_threads_completed += h_ns[i];
    }
    
    printf("n_threads_completed\t%d\n", n_threads_completed);
  #endif
    
    hipFree(arr);
    
    hipMemcpy(host_extraced_bytes, extraced_bytes, n_extracted_bytes, hipMemcpyDeviceToHost);
    hipFree(extraced_bytes);
    
}













































#include <opencv2/core/core.hpp>
#include <png.h>
#include <unistd.h> // for STD(IN|OUT)_FILENO
#include <cstdlib> // for exit()

#if defined (__unix__) || (defined (__APPLE__) && defined (__MACH__))
    #define IS_POSIX
#endif

#ifdef DEBUG
    #include <iostream>

    #ifdef IS_POSIX
        #include <execinfo.h> // for printing stack trace
    #endif
#endif

#include "config.h" // for N_BITPLANES, N_CHANNELS


void handler(int sgnl){
  #if (defined (DEBUG)) && defined (IS_POSIX)
    void* arr[10];
    
    size_t size = backtrace(arr, 10);
    
    fprintf(stderr, "E(%d):\n", sgnl);
    backtrace_symbols_fd(arr, size, STDERR_FILENO);
  #endif
    exit(sgnl);
}



class BPCSStreamBuf {
    // Based on excellent post by krzysztoftomaszewski
    // src https://artofcode.wordpress.com/2010/12/12/deriving-from-stdstreambuf/
  public:
    /* Constructors */
    BPCSStreamBuf(const uint8_t min_complexity, int img_n, int n_imgs, char** im_fps):
                // WARNING: img_fps is just argv which needs an index to skip the options
                // Use double pointer rather than array of pointers due to constraints on constructor initialisation
    not_exhausted(true),
    min_complexity(min_complexity), img_n(img_n), img_n_offset(img_n), n_imgs(n_imgs), img_fps(im_fps)
    {}
    
    
    bool not_exhausted;
    int n_extracted_bytes;
    
    uchar* img_data;
    uint8_t* extraction_byte_tensor;
    
    void gets();
    void load_next_img(); // Init
  private:
    const uint8_t min_complexity;
    
    uint8_t channel_n;
    
    uint32_t w;
    uint32_t h;
    uint32_t n_hztl_grids;
    uint32_t n_vtcl_grids;
    
    const int img_n_offset;
    int img_n;
    int n_imgs;
    
    char** img_fps;
    
    cv::Mat im_mat;
    std::vector<cv::Mat> channel_byteplanes;
};

void BPCSStreamBuf::gets(){
    if (this->channel_n == N_CHANNELS){
        if (this->img_n == this->n_imgs){
            this->not_exhausted = false;
            return;
        }
        this->load_next_img();
    }
    
  #ifdef DEBUG
    printf("Extracting channel with settings:\n\tthreshold=%d\tw=%d\th=%d\tn_hztl_grids=%d\tn_vtcl_grids=%d\n", this->min_complexity, this->w, this->h, this->n_hztl_grids, this->n_vtcl_grids);
  #endif
    
    extract_bytes(this->min_complexity, this->w, this->h, this->n_hztl_grids, this->n_vtcl_grids, this->channel_byteplanes[0].data, this->extraction_byte_tensor);
    
    this->n_extracted_bytes = 0;
    for (int j=0; j<this->n_vtcl_grids; ++j)
        for (int i=0; i<this->n_hztl_grids; ++i)
            if (this->extraction_byte_tensor[11*(this->n_hztl_grids*j + i) + 0] != 0)
                for (int k=1; k<11; ++k)
                    this->extraction_byte_tensor[this->n_extracted_bytes++] = this->extraction_byte_tensor[11*(w*j + i) + k];
                    // Index of LHS is not greater than index of RHS - this is overwriting from the 'left'
    
    --this->n_extracted_bytes;
  #ifdef DEBUG
    printf("Extracted %d bytes\n", this->n_extracted_bytes);
  #endif
    
    ++this->channel_n;
}

void BPCSStreamBuf::load_next_img(){
    if (this->img_n != this->img_n_offset){
        free(this->extraction_byte_tensor);
    }
    
  #ifdef DEBUG
    printf("Loading img %d of %d: %s\n", this->img_n, this->n_imgs, this->img_fps[this->img_n]);
  #endif
    
    /* Load PNG file into array */
    FILE* png_file = fopen(this->img_fps[this->img_n], "rb");
    
    uchar png_sig[8];
    
    fread(png_sig, 1, 8, png_file);
    if (!png_check_sig(png_sig, 8)){
        #ifdef DEBUG
        std::cerr << "Bad signature in file `" << this->img_fps[this->img_n] << "`" << std::endl;
        #endif
        handler(60);
    }
    
    auto png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    
    if (!png_ptr)
        // Could not allocate memory
        handler(4);
  
    auto png_info_ptr = png_create_info_struct(png_ptr);
    
    if (!png_info_ptr){
        png_destroy_read_struct(&png_ptr, NULL, NULL);
        handler(69);
    }
    
    /* ERROR - incorrect use of incomplete type
    if (setjmp(png_ptr->jmpbuf)){
        png_destroy_read_struct(&png_ptr, &png_info_ptr, NULL);
        handler(1);
    }
    */
    
    png_init_io(png_ptr, png_file);
    png_set_sig_bytes(png_ptr, 8);
    png_read_info(png_ptr, png_info_ptr);
    
  #ifdef TESTS
    int32_t bit_depth;
    int32_t colour_type;
  #endif
    
    png_get_IHDR(
        png_ptr, png_info_ptr, &this->w, &this->h
      #ifdef TESTS
        , &bit_depth, &colour_type
      #else
        , nullptr, nullptr
      #endif
        , NULL, NULL, NULL
    );
    
    this->n_hztl_grids = this->w/9;
    this->n_vtcl_grids = this->h/9;
    
    #ifdef TESTS
        assert(bit_depth == N_BITPLANES);
        if (colour_type != PNG_COLOR_TYPE_RGB){
            handler(61);
        }
    #endif
    
    png_read_update_info(png_ptr, png_info_ptr);
    
    #ifdef TESTS
        assert(png_get_channels(png_ptr, png_info_ptr) == 3);
        assert(png_get_rowbytes(png_ptr, png_info_ptr) == N_CHANNELS*this->w);
    #endif
    
    this->img_data = (uchar*)malloc(N_CHANNELS*this->w*this->h);
    
    uchar* row_ptrs[h];
    for (uint32_t i=0; i<this->h; ++i)
        row_ptrs[i] = this->img_data + i*N_CHANNELS*this->w;
    
    png_read_image(png_ptr, row_ptrs);
    
    fclose(png_file);
    png_destroy_read_struct(&png_ptr, &png_info_ptr, NULL);
    
    rgb2cgc(N_CHANNELS*this->w*this->h, this->img_data);
    
    this->im_mat = cv::Mat(this->h, this->w, CV_8UC3, this->img_data);
    // WARNING: Loaded as RGB rather than OpenCV's default BGR
    
    cv::split(this->im_mat, this->channel_byteplanes);
    
    free(this->img_data);
    
    this->extraction_byte_tensor = (uint8_t*)malloc(this->n_hztl_grids*this->n_vtcl_grids*11*sizeof(uint8_t));
    
    this->channel_n = 0;
    
    ++this->img_n;
}


int main(const int argc, char* argv[]){
    int i = 0;
    
  #ifdef DEBUG
    bool print_content = true;
    bool ignore_errors = false;
    int verbosity = 3;
    
    while (++i < argc){
        if (argv[i][0] == '-' && argv[i][2] == 0){
            switch(argv[i][1]){
                case 'v': ++verbosity; break;
                case 'q': --verbosity; break;
                case 'Q': print_content=false; break;
                case 'i': ignore_errors=true; break;
                default: --i; goto end_args;
            }
        } else {
            --i;
            goto end_args;
        }
    }
    end_args:
    
    if (verbosity < 0)
        verbosity = 0;
    else if (verbosity > 10)
        verbosity = 10;
  #endif
    
    const uint8_t min_complexity = 50 + (argv[++i][0] -48);
    
    BPCSStreamBuf bpcs_stream(min_complexity, ++i, argc, argv);
    bpcs_stream.load_next_img(); // Init
    
    do {
        bpcs_stream.gets();
      #ifdef DEBUG
        if (print_content)
      #endif
            write(STDOUT_FILENO, bpcs_stream.extraction_byte_tensor, bpcs_stream.n_extracted_bytes);
    } while (bpcs_stream.not_exhausted);
    free(bpcs_stream.extraction_byte_tensor);
}

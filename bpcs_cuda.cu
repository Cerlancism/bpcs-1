#include "hip/hip_runtime.h"
#include "units.h" // for uintN_t typedefs
#include "config.h" // for N_BITPLANES, N_CHANNELS
#define SET_THREAD_COORD(row, x) int row = (blockIdx.y * blockDim.y) + threadIdx.y
#define SET_THREAD_COORDS() SET_THREAD_COORD(row, x); SET_THREAD_COORD(col, y)

/* See https://en.wikipedia.org/wiki/CUDA for variious hardware-specific constraints. */


/* Device */

__global__ void gpu_rgb2cgc(int w, int h, uint8_t* arr){
    /*
        Inputs:
            row_ptrs:
                Matrix of 8-bit pixels
    */
    //int row = (blockIdx.y * blockDim.y) + threadIdx.y;
    //int col = (blockIdx.x * blockDim.x) + threadIdx.x;
    SET_THREAD_COORDS();
    if (row < h && col < w){
        // Order is because most images are landscape, therefore - and this is just a totally uneducated guess - I would expect the height condition to fail more often
        arr[col*row] ^= (arr[col*row] >> 1);
    }
}
/*
inline __device__ uint8_t gpu_calculate_grid_complexity(uint8_t bitgrid[81]){
    uint8_t c = 0;
    for (int i=0;  i < 9*9 - 1;  ++i)
        if (i % 9 != 9 - 1)
            c += bitgrid[i] ^ bitgrid[i+1];
    for (int i=0;  i < 9;  ++i)
        for (int j=0;  j < 8;  ++j)
            c += bitgrid[9*j + i] ^ bitgrid[9*(j+1) + i];
    return c;
}
*/
/*
inline __device__ void gpu_conjugate_grid(int row, int col, uint8_t** row_ptrs){
    for (int i=0; i<8; ++i)
        row_ptrs[row][col] &= (row + col) & 1;
}
*/
inline __device__ uint8_t gpu_calculate_grid_complexity(int row, int col, uint32_t w, uint8_t*& arr){
    uint8_t c = 0;
    for (int j=0; j<8; ++j)
        for (int i=0; i<9; ++i)
            c  +=  arr[w*(row + j) + (col + i)]  ^  arr[w*(row + j + 1) + (col + i)];
    for (int j=0; j<9; ++j)
        for (int i=0; i<8; ++i)
            c  +=  arr[w*(row + j) + (col + i)]  ^  arr[w*(row + j)     + (col + i + 1)];
    return c;
}

__global__ void gpu_extract_bytes(uint8_t t, uint32_t w, uint32_t h, uint32_t n_hztl_grids, uint32_t n_vtcl_grids, uint8_t*& arr, uint8_t*& extraced_bytes){
    /*
        Inputs:
            t
                aka complexity_threshold
            arr
                Channel (i.e. byteplane) matrix, flattened
    */
    SET_THREAD_COORD(row, y);
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;
    
    if (col < w/9 && row < h){
        for (int j=0; j<N_BITPLANES; ++j){
            
            if (gpu_calculate_grid_complexity(row, col, w, arr) >= t){
                extraced_bytes[11*(w*row + col) + 0] = 1;
                for (int i=1; i<11; ++i){
                    extraced_bytes[11*(w*row + col) + i] = 0;
                    for (int k=0; k<8; ++k){
                        extraced_bytes[11*(w*row + col) + i] *= 2;
                        extraced_bytes[11*(w*row + col) + i] |= (arr[w*row + col + k] >> j) & 1;
                    }
                }
                
            }
        }
    }
}



/* Host */

void rgb2cgc(uint32_t w, uint32_t h, uint8_t*& host_img_data){
    int n_grids;
    int n_blocks;
    //hipOccupancyMaxPotentialBlockSize(&n_grids, &n_blocks, gpu_rgb2cgc, 0, w*h);
    n_grids=1; n_blocks=1024;
    
    uint8_t* arr;
    hipMalloc(&arr, w*h*sizeof(uint8_t));
    hipMemcpy(arr, host_img_data, h*w*sizeof(uint8_t), hipMemcpyHostToDevice);
    
    gpu_rgb2cgc<<<n_grids, n_blocks>>>(w, h, arr);
    hipDeviceSynchronize();
    
    hipMemcpy(host_img_data, arr, h*w*sizeof(uint8_t), hipMemcpyDeviceToHost);
    hipFree(arr);
}

void extract_bytes(uint8_t t, uint32_t w, uint32_t h, uint32_t n_hztl_grids, uint32_t n_vtcl_grids, uint8_t*& host_img_data, uint8_t*& host_extraced_bytes){
    /*
        Inputs:
            host_extraced_bytes
                Empty array to write to
    */
    int n_grids;
    int n_blocks;
    //hipOccupancyMaxPotentialBlockSize(&n_grids, &n_blocks, gpu_rgb2cgc, 0, n_hztl_grids*n_vtcl_grids);
    n_grids=1; n_blocks=1024;
    
    uint8_t* arr;
    hipMalloc(&arr, w*h*sizeof(uint8_t));
    hipMemcpy(arr, host_img_data, h*w*sizeof(uint8_t), hipMemcpyHostToDevice);
    
    auto n_extracted_bytes = n_hztl_grids*n_vtcl_grids*11*sizeof(uint8_t);
    uint8_t* extraced_bytes;
    hipMalloc(&extraced_bytes, n_extracted_bytes);
    
    gpu_extract_bytes<<<n_grids, n_blocks>>>(t, w, h, n_hztl_grids, n_vtcl_grids, arr, extraced_bytes);
    hipDeviceSynchronize();
    
    hipFree(arr);
    
    hipMemcpy(host_extraced_bytes, extraced_bytes, n_extracted_bytes, hipMemcpyDeviceToHost);
    hipFree(extraced_bytes);
    
}













































#include <opencv2/core/core.hpp>
#include <png.h>
#include <unistd.h> // for STD(IN|OUT)_FILENO
#include <cstdlib> // for exit()

#if defined (__unix__) || (defined (__APPLE__) && defined (__MACH__))
    #define IS_POSIX
#endif

#ifdef DEBUG
    #include <iostream>

    #ifdef IS_POSIX
        #include <execinfo.h> // for printing stack trace
    #endif
#endif

#include "config.h" // for N_BITPLANES, N_CHANNELS


void handler(int sgnl){
  #if (defined (DEBUG)) && defined (IS_POSIX)
    void* arr[10];
    
    size_t size = backtrace(arr, 10);
    
    fprintf(stderr, "E(%d):\n", sgnl);
    backtrace_symbols_fd(arr, size, STDERR_FILENO);
  #endif
    exit(sgnl);
}



class BPCSStreamBuf {
    // Based on excellent post by krzysztoftomaszewski
    // src https://artofcode.wordpress.com/2010/12/12/deriving-from-stdstreambuf/
  public:
    /* Constructors */
    BPCSStreamBuf(const uint8_t min_complexity, int img_n, int n_imgs, char** im_fps):
                // WARNING: img_fps is just argv which needs an index to skip the options
                // Use double pointer rather than array of pointers due to constraints on constructor initialisation
    not_exhausted(true),
    min_complexity(min_complexity), img_n(img_n), img_n_offset(img_n), n_imgs(n_imgs), img_fps(im_fps)
    {}
    
    
    bool not_exhausted;
    int n_extracted_bytes;
    
    uchar* img_data;
    uint8_t* extraction_byte_tensor;
    
    void gets();
    void load_next_img(); // Init
  private:
    const uint8_t min_complexity;
    
    uint8_t channel_n;
    
    uint32_t w;
    uint32_t h;
    uint32_t n_hztl_grids;
    uint32_t n_vtcl_grids;
    
    uint32_t rowbytes;
    
    const int img_n_offset;
    int img_n;
    int n_imgs;
    
    char** img_fps;
    
    cv::Mat im_mat;
    std::vector<cv::Mat> channel_byteplanes;
};

void BPCSStreamBuf::gets(){
    if (this->channel_n == N_CHANNELS){
        if (this->img_n == this->n_imgs){
            this->not_exhausted = false;
            return;
        }
        this->load_next_img();
    }
    
    extract_bytes(this->min_complexity, this->w, this->h, this->n_hztl_grids, this->n_vtcl_grids, this->channel_byteplanes[0].data, this->extraction_byte_tensor);
    
    this->n_extracted_bytes = 0;
    for (int j=0; j<this->n_vtcl_grids; ++j)
        for (int i=0; i<this->n_hztl_grids; ++i)
            if (this->extraction_byte_tensor[11*(this->n_hztl_grids*j + i) + 0] != 0)
                for (int k=1; k<11; ++k)
                    this->extraction_byte_tensor[this->n_extracted_bytes++] = this->extraction_byte_tensor[11*(w*j + i) + k];
                    // Index of LHS is not greater than index of RHS - this is overwriting from the 'left'
    
    ++this->channel_n;
}

void BPCSStreamBuf::load_next_img(){
    if (this->img_n != this->img_n_offset){
        free(this->extraction_byte_tensor);
    }
    
    /* Load PNG file into array */
    FILE* png_file = fopen(this->img_fps[this->img_n], "rb");
    
    uchar png_sig[8];
    
    fread(png_sig, 1, 8, png_file);
    if (!png_check_sig(png_sig, 8)){
        #ifdef DEBUG
        std::cerr << "Bad signature in file `" << this->img_fps[this->img_n] << "`" << std::endl;
        #endif
        handler(60);
    }
    
    auto png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    
    if (!png_ptr)
        // Could not allocate memory
        handler(4);
  
    auto png_info_ptr = png_create_info_struct(png_ptr);
    
    if (!png_info_ptr){
        png_destroy_read_struct(&png_ptr, NULL, NULL);
        handler(69);
    }
    
    /* ERROR - incorrect use of incomplete type
    if (setjmp(png_ptr->jmpbuf)){
        png_destroy_read_struct(&png_ptr, &png_info_ptr, NULL);
        handler(1);
    }
    */
    
    png_init_io(png_ptr, png_file);
    png_set_sig_bytes(png_ptr, 8);
    png_read_info(png_ptr, png_info_ptr);
    
  #ifdef TESTS
    int32_t bit_depth;
    int32_t colour_type;
  #endif
    
    png_get_IHDR(
        png_ptr, png_info_ptr, &this->w, &this->h
      #ifdef TESTS
        , &bit_depth, &colour_type
      #else
        , nullptr, nullptr
      #endif
        , NULL, NULL, NULL
    );
    
    this->n_hztl_grids = this->w/9;
    this->n_vtcl_grids = this->h/9;
    
    #ifdef TESTS
        assert(bit_depth == N_BITPLANES);
        if (colour_type != PNG_COLOR_TYPE_RGB){
            handler(61);
        }
    #endif
    
    png_read_update_info(png_ptr, png_info_ptr);
    
    this->rowbytes = png_get_rowbytes(png_ptr, png_info_ptr);
    
    #ifdef TESTS
        assert(png_get_channels(png_ptr, png_info_ptr) == 3);
    #endif
    
    this->img_data = (uchar*)malloc(this->rowbytes*this->h);
    
    uchar* row_ptrs[h];
    for (uint32_t i=0; i<this->h; ++i)
        row_ptrs[i] = this->img_data + i*this->rowbytes;
    
    png_read_image(png_ptr, row_ptrs);
    
    fclose(png_file);
    png_destroy_read_struct(&png_ptr, &png_info_ptr, NULL);
    
    rgb2cgc(this->w, N_CHANNELS*this->h, row_ptrs[0]);
    //                == this->rowbytes
    
    this->im_mat = cv::Mat(this->h, this->w, CV_8UC3, this->img_data);
    // WARNING: Loaded as RGB rather than OpenCV's default BGR
    
    cv::split(this->im_mat, this->channel_byteplanes);
    
    free(this->img_data);
    
    this->extraction_byte_tensor = (uint8_t*)malloc(this->n_hztl_grids*this->n_vtcl_grids*11*sizeof(uint8_t));
    
    this->channel_n = 0;
    
    ++this->img_n;
}


int main(const int argc, char* argv[]){
    int i = 0;
    
    const uint8_t min_complexity = 50 + (argv[++i][0] -48);
    
    BPCSStreamBuf bpcs_stream(min_complexity, ++i, argc, argv);
    bpcs_stream.load_next_img(); // Init
    
    do {
        bpcs_stream.gets();
        write(STDOUT_FILENO, bpcs_stream.extraction_byte_tensor, bpcs_stream.n_extracted_bytes-1);
    } while (bpcs_stream.not_exhausted);
    free(bpcs_stream.extraction_byte_tensor);
}
